#include "hip/hip_runtime.h"
#include <stdio.h>

#include "gpu_routines.h"

#include <hip/hip_runtime.h>

#define BLOCK 32
#define N_COL 20

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void solveCollisions_Kernel(Circle * circles, int n, 
	Collision * colls, int * n_cols, int iterations, float gravity, float dt) 
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if(idx < (*n_cols)){
		Collision c = colls[idx];
		Circle A = circles[c.A];
		Circle B = circles[c.B];
		//printf("A(%f,%f), B(%f,%f)\n", A.px, A.py, B.px, B.py);
		float Avx = A.vx, Avy = A.vy, AvA = A.angularVelocity;
		float Bvx = B.vx, Bvy = B.vy, BvA = B.angularVelocity;
		for (int i = 0; i < iterations; ++i){
			if(A.mass + B.mass < EPS) {
		    	Avx = 0.0f;
		    	Avy = 0.0f;
		    	Bvx = 0.0f;
		    	Bvy = 0.0f;
		    	continue;
		  	}

		  	float rax = c.contact_x - A.px;
		  	float ray = c.contact_y - A.py;
		  	float rbx = c.contact_x - B.px;
		  	float rby = c.contact_y - B.py;

		  	//printf("Iter %d: rax=%f, ray=%f, rbx=%f, rby=%f\n", i, rax, ray, rbx, rby);

		  	float rvx = Bvx - (BvA * rby) - Avx + (AvA * ray); //ERROR POSIBLE
		  	float rvy = Bvy + (BvA * rbx) - Avy - (AvA * rax);

		  	float contact_vel = rvx * c.normal_x + rvy * c.normal_y;
		  	//printf("Iter %d: rvx=%f, rvy=%f, contact_vel=%f\n", i, rvx, rvy, contact_vel);
		    if(contact_vel > 0.0f)
		    	continue;

		  	float raCrossN = (rax * c.normal_y) - (ray * c.normal_x);
		  	float rbCrossN = (rbx * c.normal_y) - (rby * c.normal_x);

		  	float invMassSum = A.inv_mass + B.inv_mass + raCrossN*raCrossN * A.inv_inertia + rbCrossN*rbCrossN * B.inv_inertia; 
		  	
		  	float e = 0.2f;
		  	if((rvx * rvx + rvy * rvy) < ((dt * gravity * dt * gravity) + EPS))
		  		e = 0.0f;
		  	//printf("Iter %d: raCrossN=%f, rbCrossN=%f, contact_vel=%f, e=%f\n", i, raCrossN, rbCrossN, invMassSum,e);
		  	
		  	float j = -(1.0f + e) * contact_vel;
		  	j /= invMassSum;

		  	float impulse_x = c.normal_x * j;
		  	float impulse_y = c.normal_y * j;

		  	//printf("Iter %d: impulse_x=%f, impulse_y=%f, j=%f\n", i, impulse_x, impulse_y, j);

		  	Avx += A.inv_mass * (-impulse_x);
    		Avy += A.inv_mass * (-impulse_y);
    		AvA += A.inv_inertia * ((rax * (-impulse_y)) - (ray * (-impulse_x)));

		  	Bvx += B.inv_mass * (impulse_x);
    		Bvy += B.inv_mass * (impulse_y);
    		BvA += B.inv_inertia * ((rbx * (impulse_y)) - (rby * (impulse_x)));

    		//printf("Iter %d: A.vx =%f, A.vy=%f, A.aV=%f\n", i, A.vx, A.vy, A.angularVelocity);
    		//printf("Iter %d: B.vx =%f, B.vy=%f, B.aV=%f\n", i, B.vx, B.vy, B.angularVelocity);
		}
		atomicAdd(&circles[c.A].vx, (Avx - A.vx));
		atomicAdd(&circles[c.A].vy, (Avy - A.vy));
		atomicAdd(&circles[c.A].angularVelocity, (AvA - A.angularVelocity));

		atomicAdd(&circles[c.B].vx, (Bvx - B.vx));
		atomicAdd(&circles[c.B].vy, (Bvy - B.vy));
		atomicAdd(&circles[c.B].angularVelocity, (BvA - B.angularVelocity));
	}
}

__global__ void calculateContacs_Kernel(Circle * circles, int n, 
	Collision * colls, int * n_cols) 
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n){
		Circle ri = circles[i];
		for (unsigned int j = i + 1; j < n; ++j){
			Circle rj = circles[j];
			if(ri.inv_mass == 0.0f && rj.inv_mass == 0.0f)
				continue;
			Collision c;
			c.A = i;
			c.B = j;
			c.normal_x = rj.px - ri.px;
			c.normal_y = rj.py - ri.py;
			float dist = hypot(c.normal_x, c.normal_y);
			float suma_radius = ri.radius + rj.radius;
			if(dist >= suma_radius)
				continue; //Not contact
			
			if(dist <= EPS) {
				c.penetration = ri.radius;
				c.normal_x = 1.0f;
				c.normal_y = 0.0f;
				c.contact_x = ri.px;
				c.contact_y = ri.py;
			}
			else{
				c.penetration = suma_radius - dist;
				c.normal_x /= dist;
				c.normal_y /= dist;
				c.contact_x = c.normal_x * ri.radius + ri.px;
				c.contact_y = c.normal_y * ri.radius + ri.py;
			}
			int idx = atomicAdd(n_cols, 1);
			colls[idx] = c;
		}
	}
}

__global__ void integrateVelocities_Kernel(Circle * circles, int n, float gravity, float dt) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n){
		Circle c = circles[i];
		if(c.inv_mass > 0.0f){
			c.px += c.vx * dt;
			c.py += c.vy * dt;
			c.vy += gravity * (dt / 2.0f);
		}
		circles[i] = c;
	}
}

__global__ void integrateForces_Kernel(Circle * circles, int n, float gravity, float dt) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n){
		Circle c = circles[i];
		if(c.inv_mass > 0.0f)
			c.vy += gravity * (dt / 2.0f);
		circles[i] = c;
	}
}

__global__ void print_circle_kernel(Circle * o, int n) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n){
		printf("Circle[%d] in (%f, %f)\n", i, o[i].px, o[i].py);
	}
}

__global__ void init_context_kernel() {
	printf("Cuda context initialized!\n");
}

void GPU::solveCollisions_GPU(vector<Collision> &contacts){

	dim3 dimGrid(ceil((float)this->n_cols / BLOCK));
	dim3 dimBlock(BLOCK);
	solveCollisions_Kernel<<<dimGrid,dimBlock>>>(circles_GPU, this->lro->size(), 
		colls_GPU, this->n_cols_GPU, iterations, gravity, dt);
	hipDeviceSynchronize();
	/*
	contacts.resize(n_cols);
	hipMemcpy(&(contacts[0]), &colls_GPU[0], 
			sizeof(Collision) * this->n_cols, hipMemcpyDeviceToHost);

	hipFree(colls_GPU);
	hipFree(n_cols_GPU);
	*/
}

void GPU::calculateContact_GPU(vector<Collision> &contacts){
	this->n_cols = 0; 
	hipMalloc((void **) &this->n_cols_GPU, sizeof(int));
	hipMemcpy(this->n_cols_GPU, &this->n_cols, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **) &colls_GPU, sizeof(Collision) * this->lro->size() * 30);

	dim3 dimGrid(ceil((float)this->lro->size() / BLOCK));
	dim3 dimBlock(BLOCK);
	calculateContacs_Kernel<<<dimGrid,dimBlock>>>(circles_GPU, this->lro->size(), 
		colls_GPU, this->n_cols_GPU);
	hipDeviceSynchronize();

	hipMemcpy(&this->n_cols, this->n_cols_GPU, sizeof(int), hipMemcpyDeviceToHost);
	//printf("N_COLS = %d\n", n_cols);
	
	contacts.resize(n_cols);
	hipMemcpy(&(contacts[0]), &colls_GPU[0], 
			sizeof(Collision) * this->n_cols, hipMemcpyDeviceToHost);
	
	hipFree(colls_GPU);
	hipFree(n_cols_GPU);
	
}

void GPU::integrateVelocities_GPU(){
	dim3 dimGrid(ceil((float)this->lro->size() / BLOCK));
	dim3 dimBlock(BLOCK);
	integrateVelocities_Kernel<<<dimGrid,dimBlock>>>(circles_GPU, this->lro->size(), gravity, dt);
	hipDeviceSynchronize();
}

void GPU::integrateForces_GPU(){
	dim3 dimGrid(ceil((float)this->lro->size() / BLOCK));
	dim3 dimBlock(BLOCK);
	integrateForces_Kernel<<<dimGrid,dimBlock>>>(circles_GPU, this->lro->size(), gravity, dt);
	hipDeviceSynchronize();
}

void GPU::initializeContext(){
	//Initialize cuda contex
	init_context_kernel<<<1,1>>>();
	hipDeviceSynchronize();
}

void GPU::update_mem(){
	if(true || this->lro->size() > this->N_GPU_obj){
		//Hay objetos que faltan en GPU
		int init = 0;//this->N_GPU_obj;
		int num_copy = this->lro->size();// - this->N_GPU_obj;
		if(this->lro->size() > this->MAX_GPU_obj){ //Si no caben en GPU
			init = 0;
			num_copy = this->lro->size();
			this->MAX_GPU_obj = 2 * this->lro->size();

			hipFree(circles_GPU); //Liberamos la memoria existente
			hipMalloc((void **) &circles_GPU, sizeof(Circle) * this->MAX_GPU_obj);
		}

		//CPU -> GPU
		//printf("COPIADO desde %d con n = %d\n", init, num_copy);
		hipMemcpy(&circles_GPU[init], &(this->lro->vro[init]), 
			sizeof(Circle) * num_copy, hipMemcpyHostToDevice);

		this->N_GPU_obj = this->lro->size();
	}
	this->copy_DeviceToHost();
}

void GPU::copy_HostToDevice(){
	hipFree(circles_GPU);
	hipMalloc((void **) &circles_GPU, sizeof(Circle) * this->lro->size());
	hipMemcpy(&circles_GPU[0], &(this->lro->vro[0]), 
			sizeof(Circle) * this->lro->size(), hipMemcpyHostToDevice);
}

void GPU::copy_DeviceToHost(){
	hipMemcpy(&(this->lro->vro[0]), &circles_GPU[0], 
			sizeof(Circle) * this->lro->size(), hipMemcpyDeviceToHost);
}

GPU::GPU(){
	this->N_GPU_obj = 0;
	this->MAX_GPU_obj = 0;
}

GPU::GPU(ListCircles * list){
	this->lro = list;
	this->N_GPU_obj = 0;
	this->MAX_GPU_obj = 0;
}